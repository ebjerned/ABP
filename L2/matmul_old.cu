
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <vector>

#define AssertCuda(error_code) \
if(error_code != hipSuccess) 	\
{ \
	std::cout << "The cuda call in" << __FILE__ << " on line "\
	<< __LINE__ << " resulted in the error '" \
	<< hipGetErrorString(error_code) << "'" << std::endl;\
	std::abort();\
}\

const int block_size = 128;
const int chunk_size = 1;


__global__ void set_vector(const int N, const float val, float *x)
{
  const int idx_base = threadIdx.x + blockIdx.x * (blockDim.x * chunk_size);
  for (unsigned int i = 0; i < chunk_size; ++i)
    {
      const int idx = idx_base + i * block_size;
      if (idx < N)
        x[idx] = val;
    }
}

__global__ void set_vector_rising(const int N, const float val, float *x)
{
  const int idx_base = threadIdx.x + blockIdx.x * (blockDim.x * chunk_size);
  for (unsigned int i = 0; i < chunk_size; ++i)
    {
      const int idx = idx_base + i * block_size;
      if (idx < N)
        x[idx] =val*(blockIdx.x*blockDim.x+threadIdx.x + threadIdx.y*gridDim.x*blockDim.x);
		
    }
}

__device__ void warpReduce0(volatile float* sdata, int tx){
    sdata[tx] += sdata[tx + 32];
    sdata[tx] += sdata[tx + 16];
    sdata[tx] += sdata[tx + 8];
    sdata[tx] += sdata[tx + 4];
    sdata[tx] += sdata[tx + 2];
    sdata[tx] += sdata[tx + 1];

}
template <unsigned int blockSize>
__device__ void warpReduce1(volatile float* sdata, int tx){
    if(blockSize >= 64) sdata[tx] += sdata[tx +32];
    if(blockSize >= 32) sdata[tx] += sdata[tx +16];
    if(blockSize >= 16) sdata[tx] += sdata[tx +8];
    if(blockSize >= 8) sdata[tx] += sdata[tx +4];
    if(blockSize >= 4) sdata[tx] += sdata[tx +2];
    if(blockSize >= 2) sdata[tx] += sdata[tx +1];


}

__global__ void matmat(const float* A, const float* B, float* C, unsigned const int M , unsigned const int N, unsigned const K){
	int col = blockDim.x*blockIdx.x + threadIdx.x;
	int row = blockDim.y*blockIdx.y + threadIdx.y;
	
	if((col >= N)|| (row >= M)) return;


	int roof = (M+blockDim.y-1)/blockDim.y;
	roof = (row + blockDim.y*(roof-1)) < M ? roof : roof -1;
	for(unsigned int currentRow = 0; currentRow < N; ++currentRow){
		float sum = 0.f;
		for(unsigned int k= 0; k < roof; k++){
			float coeff = B[K*currentRow + col+ blockDim.y*k];
			sum += A[col*M+row + (blockDim.y*k)]*coeff;
	}
	
	__syncthreads();
	atomicAdd(&C[col+currentRow*K], sum);
   }
}

__global__ void matvec_old(const float* A, const float* B, float* C, unsigned const int M , unsigned const int N	){
	int col = blockDim.x*blockIdx.x + threadIdx.x;
	int row = blockDim.y*blockIdx.y + threadIdx.y;
	
	if((col >= N)|| (row >= M)) return;

	int roof = (M+blockDim.y-1)/blockDim.y;
	roof = (row + blockDim.y*(roof-1)) < M ? roof : roof -1;
	float sum = 0.f;
	
	float coeff = B[col];
	for(unsigned int k= 0; k < roof; k++){
		sum = A[col*M+row + (blockDim.y*k)]*coeff;    
	    //__syncthreads();
    	//atomicAdd(&C[row+blockDim.y*k], sum);
        C[row+blockDim.y*k] += sum;
	}
	

   
}

__global__ void matvec0(const float* A, const float* B, float* C, unsigned const int M , unsigned const int N	){
    __shared__ float sdata[block_size];

    int col = blockDim.x*blockIdx.x + threadIdx.x;
	int row = blockDim.y*blockIdx.y + threadIdx.y;
    int tx = threadIdx.x;
	if((col >= N)|| (row >= M)){
        sdata[tx]=0;
    }else{
        sdata[tx] = A[col*M+row]*B[col];
    }

    __syncthreads();
    for(unsigned int s = 1; s<blockDim.x; s*=2){
        if(tx %(2*s) == 0){
            sdata[tx]+=sdata[tx+s];
        }
        __syncthreads();
    
    }

    if(tx == 0) atomicAdd(&C[row], sdata[0]);

   
}

__global__ void matvec1(const float* A, const float* B, float* C, unsigned const int M , unsigned const int N	){
    __shared__ float sdata[block_size];

    int col = blockDim.x*blockIdx.x + threadIdx.x;
	int row = blockDim.y*blockIdx.y + threadIdx.y;
    int tx = threadIdx.x;
	if((col >= N)|| (row >= M)){
        sdata[tx]=0;
    }else{
        sdata[tx] = A[col*M+row]*B[col];
    }

    __syncthreads();
    for(unsigned int s = 1; s<blockDim.x; s*=2){
        int index = 2*s*tx;
        if(index < blockDim.x){
            sdata[index]+=sdata[index+s];
        }
        __syncthreads();
    
    }

    if(tx == 0) atomicAdd(&C[row], sdata[0]);

   
}

__global__ void matvec2(const float* A, const float* B, float* C, unsigned const int M , unsigned const int N	){
    __shared__ float sdata[block_size];

    int col = blockDim.x*blockIdx.x + threadIdx.x;
	int row = blockDim.y*blockIdx.y + threadIdx.y;
    int tx = threadIdx.x;
	if((col >= N)|| (row >= M)){
        sdata[tx]=0;
    }else{
        sdata[tx] = A[col*M+row]*B[col];
    }

    __syncthreads();
    for(unsigned int s = blockDim.x/2; s>0; s>>=1){
        if(tx < s){
            sdata[tx]+=sdata[tx+s];
        }
        __syncthreads();
    
    }

    if(tx == 0) atomicAdd(&C[row], sdata[0]);

   
}
__global__ void matvec3(const float* A, const float* B, float* C, unsigned const int M , unsigned const int N	){
    __shared__ float sdata[block_size];

    int col = (blockDim.x*2)*blockIdx.x + threadIdx.x;
	int row = blockDim.y*blockIdx.y + threadIdx.y;
    int tx = threadIdx.x;

    float sum = ((col >= N) || (row >= M)) ? 0 : A[col*M+row]*B[col];
    if((col + blockDim.x) < N){
        sum += A[(col+blockDim.x)*M+row]*B[col+blockDim.x];
    }
    sdata[tx] = sum;

/*
    if((col >= N)|| (row >= M) || ((col+blockDim.x) >= N)){
        sdata[tx]=0;
        //if((col+blockDim.x) >= N) printf("\t %i\n", col + blockDim.x);
    }else{
        sdata[tx] = A[col*M+row]*B[col] + A[(col+blockDim.x)*M+row]*B[col+blockDim.x];

    }
*/
    __syncthreads();
    for(unsigned int s = blockDim.x/2; s>0; s>>=1){
        if(tx < s){
            sdata[tx]+=sdata[tx+s];
        }
        __syncthreads();
    
    }

    if(tx == 0) atomicAdd(&C[row], sdata[0]);

}   
__global__ void matvec4(const float* A, const float* B, float* C, unsigned const int M , unsigned const int N	){
    __shared__ float sdata[block_size];

    int col = (blockDim.x*2)*blockIdx.x + threadIdx.x;
	int row = blockDim.y*blockIdx.y + threadIdx.y;
    int tx = threadIdx.x;

    float sum = ((col >=N) || (row >=M)) ? 0 :A[col*M+row]*B[col];
    if((col + blockDim.x) < N) sum += A[(col+blockDim.x)*M+row]*B[col+blockDim.x];
    sdata[tx] = sum;


    __syncthreads();
    for(unsigned int s = blockDim.x/2; s>32; s>>=1){
        if(tx < s){
            sdata[tx]+=sdata[tx+s];
        }
        __syncthreads();
    
    }
    if(tx < 32) warpReduce0(sdata, tx);

    if(tx == 0) atomicAdd(&C[row], sdata[0]);

}
template <unsigned int blockSize>
__global__ void matvec(const float* A, const float* B, float* C, unsigned const int M , unsigned const int N	){
    __shared__ float sdata[block_size];

    int col = (blockDim.x*2)*blockIdx.x + threadIdx.x;
	int row = blockDim.y*blockIdx.y + threadIdx.y;
    int tx = threadIdx.x;

    float sum = ((col >=N) || (row >=M)) ? 0 :A[col*M+row]*B[col];
    if((col + blockDim.x) < N) sum += A[(col+blockDim.x)*M+row]*B[col+blockDim.x];
    sdata[tx] = sum;


    __syncthreads();
/*    for(unsigned int s = blockDim.x/2; s>32; s>>=1){
        if(tx < s){
            sdata[tx]+=sdata[tx+s];
        }
        __syncthreads();
    
    }*/
    if(blockSize >= 1024){
        if(tx < 512) {sdata[tx] += sdata[tx+512];} __syncthreads();}
    if(blockSize >= 512){
        if(tx < 256) {sdata[tx] += sdata[tx+256];} __syncthreads();}
    if(blockSize >= 256){
        if(tx < 128) {sdata[tx] += sdata[tx+128];} __syncthreads();}
    if(blockSize >= 128){
        if(tx < 64) {sdata[tx] += sdata[tx+64];} __syncthreads();}
    if(tx < 32) warpReduce1<blockSize>(sdata, tx);

    if(tx == 0) atomicAdd(&C[row], sdata[0]);

}

template <unsigned int blockSize>
__global__ void matvec6(const float* A, const float* B, float* C, unsigned const int M , unsigned const int N	){
    __shared__ float sdata[block_size];

    int col = (blockDim.x*2)*blockIdx.x + threadIdx.x;
	int row = blockDim.y*blockIdx.y + threadIdx.y;
    int tx = threadIdx.x;
    int grid = blockSize*2*gridDim.x;
//    float sum = ((col >=N) || (row >=M)) ? 0 :A[col*M+row]*B[col];
//    if((col + blockDim.x) < N) sum += A[(col+blockDim.x)*M+row]*B[col+blockDim.x];
//    sdata[tx] = sum;
    sdata[tx] = 0;  
    while (col < N){
        if(row < M) sdata[tx] += A[col*M+row]*B[col];
        if((col+blockDim.x)<N) sdata[tx] += A[(col+blockDim.x)*M+row]*B[col+blockDim.x];
        col += grid;
    }

    __syncthreads();

    if(blockSize >= 1024){
        if(tx < 512) {sdata[tx] += sdata[tx+512];} __syncthreads();}
    if(blockSize >= 512){
        if(tx < 256) {sdata[tx] += sdata[tx+256];} __syncthreads();}
    if(blockSize >= 256){
        if(tx < 128) {sdata[tx] += sdata[tx+128];} __syncthreads();}
    if(blockSize >= 128){
        if(tx < 64) {sdata[tx] += sdata[tx+64];} __syncthreads();}
    if(tx < 32) warpReduce1<blockSize>(sdata, tx);

    if(tx == 0) atomicAdd(&C[row], sdata[0]);

}
__global__ void matvecT(const float* A, const float* B, float* C, unsigned const int M , unsigned const int N	){
	int col = blockDim.x*blockIdx.x + threadIdx.x;
	int row = blockDim.y*blockIdx.y + threadIdx.y;
	
	if((col >= N)|| (row >= M)) return;

	int roof = (M+blockDim.y-1)/blockDim.y;
	roof = (row + blockDim.y*(roof-1)) < M ? roof : roof -1;
	float sum = 0.f;
	float coeff = B[col];
	for(unsigned int k= 0; k < roof; k++){
		sum += A[col+row*N + (blockDim.y*k)]*coeff;
	}
	
	__syncthreads();
	atomicAdd(&C[col], sum);
   
}
void matmat_naive(const float* A, const float* B, float* C, unsigned const int M, unsigned const int N, unsigned const K){
	for(int i = 0; i < M;++i)
		for(int k = 0; k < K; ++k){
			C[i+M*k]=0.f;
			for(int j = 0; j < N; ++j)
				C[i+M*k] += A[i+j*M]*B[j+k*N];
		}

}

void matmat_naiveT(const float* A, const float* B, float* C, unsigned const int M, unsigned const int N, unsigned const K){
	for(int i = 0; i < M;++i)
		for(int k = 0; k < K; ++k){
			C[i+M*k]=0.f;
			for(int j = 0; j < N; ++j)
				C[i+M*k] += A[i*N+j]*B[j+k*N];
		}

}
void benchmark_mat(  const std::size_t M,
					 const std::size_t N,
                     const std::size_t K)
{

  hipError_t errorCode;

  float *A, *B, *C;
  errorCode = hipMalloc(&A, M * N * sizeof(float));
  AssertCuda(errorCode);
  errorCode = hipMalloc(&B, N * K * sizeof(float));
  AssertCuda(errorCode);
  errorCode = hipMalloc(&C, M * K* sizeof(float));
  AssertCuda(errorCode);


  set_vector<<<(M*N+block_size-1)/block_size, block_size>>>(M*N, 1.f/*/sqrt(N)*/, A);
 
  errorCode = hipGetLastError();
  AssertCuda(errorCode);
  set_vector<<<(N*K+block_size-1)/block_size, block_size>>>(N*K, 1.f/*/sqrt(N)*/, B);
 
  errorCode = hipGetLastError();
  AssertCuda(errorCode);
  set_vector<<<(M*K+block_size-1)/block_size, block_size>>>(M*K, 0.f, C);
  errorCode = hipGetLastError();
  AssertCuda(errorCode);

  std::vector<float> result_host(M*K);
  
  dim3 gridDim(ceil(0.5f*(float)M/(float)block_size),N);
  dim3 blockDim(block_size,1);
  
  const unsigned int n_tests = 20;
  const unsigned int n_repeat = 20;
  double best = 1e10, worst = 0, avg = 0;
  for (unsigned int t = 0; t < n_tests; ++t)
    {

 	  set_vector<<<(M*K+block_size-1)/block_size, block_size>>>(M*K, 0.f, C);
      const auto t1 = std::chrono::steady_clock::now();

      for (unsigned int rep = 0; rep < n_repeat; ++rep){
 	 //   set_vector<<<(M*K+block_size-1)/block_size, block_size>>>(M*K, 0.f, C);

//		if(K > 1){
//			matmat<<<gridDim, blockDim>>>(A, B, C, M, N, K);
//		}else{
            matvec<block_size><<<gridDim, blockDim>>>(A, B, C, M, N);
//		}
//	    errorCode = cudaGetLastError();
//	    AssertCuda(errorCode);
	  }
      hipDeviceSynchronize();

	//	matmat_naive(A, B, C, M, N, K);

      const double time =
        std::chrono::duration_cast<std::chrono::duration<double>>(
          std::chrono::steady_clock::now() - t1)
          .count();

      best  = std::min(best, time / n_repeat);
      worst = std::max(worst, time / n_repeat);
      avg += time / n_repeat;
    }

  errorCode = hipMemcpy(result_host.data(),  C, M *K* sizeof(float), hipMemcpyDeviceToHost);  
  AssertCuda(errorCode);

 //Printing for checking correctness
/* for(unsigned int i = 0; i <M*K;++i){
  	std::cout << result_host[(i*M)%(M*K)+(i/K)] << " ";
	if (i % K == K-1) std::cout << "" << std::endl;
  }*/
/*  
  for(unsigned int i = 0; i < M; ++i)
  	std::cout << result_host[i] << std::endl;
*/
  errorCode = hipFree(A);
  AssertCuda(errorCode);
  errorCode = hipFree(B);
  AssertCuda(errorCode);
  errorCode = hipFree(C);
  AssertCuda(errorCode);

  std::cout << "MATMUL (GPU) of size (M,N,K) " << std::setw(8) << M << "  " << N << " " << K 
            << " : min/avg/max: " << std::setw(11) << best << " "
            << std::setw(11) << avg / n_tests << " " << std::setw(11) << worst
            << " seconds or " << std::setw(8) << 1e-9 * 2 * N * M * N / best
            << " GFLOPS/s or " << std::setw(8)
            << 1e-9 * sizeof(float) *(N*M + M + N) / best << " GB/s" << std::endl; 

}

int main(int argc, char **argv)
{
  if (argc % 2 == 0)
    {
      std::cout << "Error, expected odd number of common line arguments"
                << std::endl
                << "Expected line of the form" << std::endl
                << "-M rows -N columns/rows -K columns" << std::endl;
      std::abort();
    }

  long M  = 8;
  long N  = -1;
  long K = 1;
  // parse from the command line
  for (int l = 1; l < argc; l += 2)
    {
      std::string option = argv[l];
      if (option == "-M")
        M = static_cast<long>(std::stod(argv[l + 1]));
      else if (option == "-N")
        N = static_cast<long>(std::stod(argv[l + 1]));
      else if (option == "-K")
        K = static_cast<long>(std::stod(argv[l + 1]));
      else
        std::cout << "Unknown option " << option << " - ignored!" << std::endl;
    }
  if(N < 0) N = M;

  //For running series test
for(float i = 7; i < 14; i+= 0.2){
  		long size = round(pow(2,i));
		benchmark_mat(size,size,K);
  }


// benchmark_mat(M, N, K);

  return 0;
}
