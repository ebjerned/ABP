
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <vector>

#define AssertCuda(error_code) \
if(error_code != hipSuccess) 	\
{ \
	std::cout << "The cuda call in" << __FILE__ << " on line "\
	<< __LINE__ << " resulted in the error '" \
	<< hipGetErrorString(error_code) << "'" << std::endl;\
	std::abort();\
}\

const int block_size = 128;
const int chunk_size = 1;


__global__ void set_vector(const int N, const float val, float *x)
{
  const int idx_base = threadIdx.x + blockIdx.x * (blockDim.x * chunk_size);
  for (unsigned int i = 0; i < chunk_size; ++i)
    {
      const int idx = idx_base + i * block_size;
      if (idx < N)
        x[idx] = val;
    }
}

__global__ void set_vector_rising(const int N, const float val, float *x)
{
  const int idx_base = threadIdx.x + blockIdx.x * (blockDim.x * chunk_size);
  for (unsigned int i = 0; i < chunk_size; ++i)
    {
      const int idx = idx_base + i * block_size;
      if (idx < N)
        x[idx] =val*(blockIdx.x*blockDim.x+threadIdx.x + threadIdx.y*gridDim.x*blockDim.x);
		
    }
}

__global__ void matmat(const float* A, const float* B, float* C, unsigned const int M , unsigned const int N, unsigned const K){
	int col = blockDim.x*blockIdx.x + threadIdx.x;
	int row = blockDim.y*blockIdx.y + threadIdx.y;
	
	if((col >= N)|| (row >= M)) return;


	int roof = (M+blockDim.y-1)/blockDim.y;
	roof = (row + blockDim.y*(roof-1)) < M ? roof : roof -1;
	for(unsigned int currentRow = 0; currentRow < N; ++currentRow){
		float sum = 0.f;
		for(unsigned int k= 0; k < roof; k++){
			float coeff = B[K*currentRow + col+ blockDim.y*k];
			sum += A[col*M+row + (blockDim.y*k)]*coeff;
	}
	
	__syncthreads();
	atomicAdd(&C[col+currentRow*K], sum);
   }
}

__global__ void matvec(const float* A, const float* B, float* C, unsigned const int M , unsigned const int N	){
	int col = blockDim.x*blockIdx.x + threadIdx.x;
	int row = blockDim.y*blockIdx.y + threadIdx.y;
	
	if((col >= N)|| (row >= M)) return;

	int roof = (M+blockDim.y-1)/blockDim.y;
	roof = (row + blockDim.y*(roof-1)) < M ? roof : roof -1;
	float sum = 0.f;
	
	float coeff = B[col];
	for(unsigned int k= 0; k < roof; k++){
		sum += A[col*M+row + (blockDim.y*k)]*coeff;
	}
	
	__syncthreads();
	atomicAdd(&C[col], sum);
   
}


__global__ void matvecT(const float* A, const float* B, float* C, unsigned const int M , unsigned const int N	){
	int col = blockDim.x*blockIdx.x + threadIdx.x;
	int row = blockDim.y*blockIdx.y + threadIdx.y;
	
	if((col >= N)|| (row >= M)) return;

	int roof = (M+blockDim.y-1)/blockDim.y;
	roof = (row + blockDim.y*(roof-1)) < M ? roof : roof -1;
	float sum = 0.f;
	float coeff = B[col];
	for(unsigned int k= 0; k < roof; k++){
		sum += A[col+row*N + (blockDim.y*k)]*coeff;
	}
	
	__syncthreads();
	atomicAdd(&C[col], sum);
   
}
void matmat_naive(const float* A, const float* B, float* C, unsigned const int M, unsigned const int N, unsigned const K){
	for(int i = 0; i < M;++i)
		for(int k = 0; k < K; ++k){
			C[i+M*k]=0.f;
			for(int j = 0; j < N; ++j)
				C[i+M*k] += A[i+j*M]*B[j+k*N];
		}

}

void matmat_naiveT(const float* A, const float* B, float* C, unsigned const int M, unsigned const int N, unsigned const K){
	for(int i = 0; i < M;++i)
		for(int k = 0; k < K; ++k){
			C[i+M*k]=0.f;
			for(int j = 0; j < N; ++j)
				C[i+M*k] += A[i*N+j]*B[j+k*N];
		}

}
void benchmark_mat(  const std::size_t M,
					 const std::size_t N,
                     const std::size_t K)
{

  hipError_t errorCode;

  float *A, *B, *C;
  errorCode = hipMalloc(&A, M * N * sizeof(float));
  AssertCuda(errorCode);
  errorCode = hipMalloc(&B, N * K * sizeof(float));
  AssertCuda(errorCode);
  errorCode = hipMalloc(&C, M * K* sizeof(float));
  AssertCuda(errorCode);


  set_vector<<<(M*N+block_size-1)/block_size, block_size>>>(M*N, 1.f/sqrt(N), A);
 
  errorCode = hipGetLastError();
  AssertCuda(errorCode);
  set_vector<<<(N*K+block_size-1)/block_size, block_size>>>(N*K, 1.f/sqrt(N), B);
 
  errorCode = hipGetLastError();
  AssertCuda(errorCode);
  set_vector<<<(M*K+block_size-1)/block_size, block_size>>>(M*K, 0.f, C);
  errorCode = hipGetLastError();
  AssertCuda(errorCode);

  std::vector<float> result_host(M*K);
  
  dim3 gridDim(N,1);
  dim3 blockDim(1,block_size);
  
  const unsigned int n_tests = 20;
  const unsigned int n_repeat = 1;
  double best = 1e10, worst = 0, avg = 0;
  for (unsigned int t = 0; t < n_tests; ++t)
    {
      const auto t1 = std::chrono::steady_clock::now();

      for (unsigned int rep = 0; rep < n_repeat; ++rep){
  		set_vector<<<(M*K+block_size-1)/block_size, block_size>>>(M*K, 0.f, C);
		if(K > 1){
			matmat<<<gridDim, blockDim>>>(A, B, C, M, N, K);
		}else{
			matvec<<<gridDim, blockDim>>>(A, B, C, M, N);
		}
	    errorCode = hipGetLastError();
  	    AssertCuda(errorCode);
	  }
      hipDeviceSynchronize();

	//	matmat_naive(A, B, C, M, N, K);

      const double time =
        std::chrono::duration_cast<std::chrono::duration<double>>(
          std::chrono::steady_clock::now() - t1)
          .count();

      best  = std::min(best, time / n_repeat);
      worst = std::max(worst, time / n_repeat);
      avg += time / n_repeat;
    }

  errorCode = hipMemcpy(result_host.data(),  C, M *K* sizeof(float), hipMemcpyDeviceToHost);  
  AssertCuda(errorCode);

/* //Printing for checking correctness
 for(unsigned int i = 0; i <M*K;++i){
  	std::cout << result_host[(i*M)%(M*K)+(i/K)] << " ";
	if (i % K == K-1) std::cout << "" << std::endl;
  }
  
  for(unsigned int i = 0; i < M; ++i)
  	std::cout << result_host[i] << std::endl;
*/
  errorCode = hipFree(A);
  AssertCuda(errorCode);
  errorCode = hipFree(B);
  AssertCuda(errorCode);
  errorCode = hipFree(C);
  AssertCuda(errorCode);

  std::cout << "MATMUL (GPU) of size (M,N,K) " << std::setw(8) << M << "  " << N << " " << K 
            << " : min/avg/max: " << std::setw(11) << best << " "
            << std::setw(11) << avg / n_tests << " " << std::setw(11) << worst
            << " seconds or " << std::setw(8) << 1e-9 * 2 * N * M * N / best
            << " GFLOPS/s or " << std::setw(8)
            << 1e-9 * sizeof(float) *(N*M + M + N) / best << " GB/s" << std::endl; 

}

int main(int argc, char **argv)
{
  if (argc % 2 == 0)
    {
      std::cout << "Error, expected odd number of common line arguments"
                << std::endl
                << "Expected line of the form" << std::endl
                << "-M rows -N columns/rows -K columns" << std::endl;
      std::abort();
    }

  long M  = 8;
  long N  = -1;
  long K = 1;
  // parse from the command line
  for (int l = 1; l < argc; l += 2)
    {
      std::string option = argv[l];
      if (option == "-M")
        M = static_cast<long>(std::stod(argv[l + 1]));
      else if (option == "-N")
        N = static_cast<long>(std::stod(argv[l + 1]));
      else if (option == "-K")
        K = static_cast<long>(std::stod(argv[l + 1]));
      else
        std::cout << "Unknown option " << option << " - ignored!" << std::endl;
    }
  if(N < 0) N = M;

  //For running series test
/*for(float i = 7; i < 14; i+= 0.2){
  		long size = round(pow(2,i));
		benchmark_mat(size,size,K);
  }*/


 benchmark_mat(M, N, K);

  return 0;
}
