

#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <vector>

#define AssertCuda(error_code) \
if(error_code != hipSuccess) 	\
{ \
	std::cout << "The cuda call in" << __FILE__ << " on line "\
	<< __LINE__ << " resulted in the error '" \
	<< hipGetErrorString(error_code) << "'" << std::endl;\
	std::abort();\
}\




const int block_size = 64;
const int chunk_size = 1;

__global__ void compute_triad(const int    N,
                              const float  a,
                              const float *x,
                              const float *y,
                              float *      z)
{
  const int idx_base = threadIdx.x + blockIdx.x * (blockDim.x * chunk_size);
  for (unsigned int i = 0; i < chunk_size; ++i)
    {
      const int idx = idx_base + i * block_size;
      if (idx < N)
        z[idx] = a * x[idx] + y[idx];
    }
}


__global__ void set_vector(const int N, const float val, float *x)
{
  const int idx_base = threadIdx.x + blockIdx.x * (blockDim.x * chunk_size);
  for (unsigned int i = 0; i < chunk_size; ++i)
    {
      const int idx = idx_base + i * block_size;
      if (idx < N)
        x[idx] = val;
    }
}

__global__ void set_vector_rising(const int N, const float val, float *x)
{
  const int idx_base = threadIdx.x + blockIdx.x * (blockDim.x * chunk_size);
  for (unsigned int i = 0; i < chunk_size; ++i)
    {
      const int idx = idx_base + i * block_size;
      if (idx < N)
        x[idx] = val*(blockIdx.x*blockDim.x+threadIdx.x + threadIdx.y*gridDim.x*blockDim.x);
    }
}



__global__ void matmul(const float*A, const float*B, float* C, unsigned const int M, unsigned const int N){
	size_t i = blockIdx.y*blockDim.y + threadIdx.y;
	size_t j = blockIdx.x*blockDim.x + threadIdx.x;

	if((i>= M) || (j >= N)){
		return;
	}

	float acc_sum = 0;
	for(unsigned int k = 0; k<N; ++k)
	{
		acc_sum += A[i+k*N]*B[k+j*N];
	}
//	__syncthreads();
	
	C[i*N+j] = acc_sum;

}

__global__ void matvec(const float* A, const float* x, float* b, unsigned const int M, unsigned const int N){
	int index = blockDim.x*blockIdx.x + threadIdx.x;
	float sum = 0.f;
	for(unsigned int k = 0; k < N; ++k)
		sum += A[N*index + k]*x[k];

	b[index] = sum;
//	__syncthreads();
//	atomicAdd(&b[index],sum);
}



// Run the actual benchmark
void benchmark_triad(const bool        align,
                     const std::size_t N,
                     const long long   repeat)
{

  unsigned int elementsSidePerBlock = 1;
  dim3 blockDimensions(ceil(N/elementsSidePerBlock),ceil(N/elementsSidePerBlock));
  hipError_t errorCode;

  float *v1, *v2, *v3;
  // allocate memory on the device
  errorCode = hipMalloc(&v1, N * N * sizeof(float));
  AssertCuda(errorCode);
  errorCode = hipMalloc(&v2, N * sizeof(float));
  AssertCuda(errorCode);
  errorCode = hipMalloc(&v3, N * sizeof(float));
  AssertCuda(errorCode);

  const unsigned int n_blocks = (N + block_size - 1) / block_size;

  set_vector<<<blockDimensions, block_size>>>(N*N, 1.f, v1);
  errorCode = hipGetLastError();
  AssertCuda(errorCode);
  set_vector<<<blockDimensions, block_size>>>(N, 1.f, v2);
  errorCode = hipGetLastError();
  AssertCuda(errorCode);
  set_vector<<<blockDimensions, block_size>>>(N, 0.f, v3);
  errorCode = hipGetLastError();
  AssertCuda(errorCode);

  std::vector<float> result_host(N);
  dim3 gridDim(ceil(N/block_size));
  dim3 blockDim(block_size);
  const unsigned int           n_tests = 20;
  /*const unsigned long long int n_repeat =
    repeat > 0 ? repeat : std::max(1UL, 100000000U / N);*/
  const unsigned int n_repeat = 1;
  double best = 1e10, worst = 0, avg = 0;
  for (unsigned int t = 0; t < n_tests; ++t)
    {
      // type of t1: std::chrono::steady_clock::time_point
      const auto t1 = std::chrono::steady_clock::now();

      for (unsigned int rep = 0; rep < n_repeat; ++rep)
      //matmul<<<blockDimensions, block_size>>>(v1, v2, v3, N, N);
      matvec<<<gridDim, blockDim>>>(v1, v2, v3, N, N);
	  errorCode = hipGetLastError();
  	  AssertCuda(errorCode);
      hipDeviceSynchronize();
      // measure the time by taking the difference between the time point
      // before starting and now
      const double time =
        std::chrono::duration_cast<std::chrono::duration<double>>(
          std::chrono::steady_clock::now() - t1)
          .count();

      best  = std::min(best, time / n_repeat);
      worst = std::max(worst, time / n_repeat);
      avg += time / n_repeat;
    }

  // Copy the result back to the host
  errorCode = hipMemcpy(result_host.data(), v1, N*N * sizeof(float), hipMemcpyDeviceToHost);  
  AssertCuda(errorCode);


 for(unsigned int i = 0; i <N*N;++i){
  	std::cout << result_host[(i*N)%(N*N)+(i/N)] << " ";
	if (i % N == N-1) std::cout << "" << std::endl;
  }
  
/*  for(unsigned int i = 0; i < N; ++i)
  	std::cout << result_host[i] << std::endl;*/
  //Not perfect check for correctness, works for 8 but not for 512 or larger
  if (result_host[0] != N*((N-1)*N*(2*N-1)/6))
    std::cout << "Error in computation, got "
              << (result_host[0] + result_host[N - 1]) << " instead of 526"
              << std::endl;

  // Free the memory on the device
  errorCode = hipFree(v1);
  AssertCuda(errorCode);
  errorCode = hipFree(v2);
  AssertCuda(errorCode);
  errorCode = hipFree(v3);
  AssertCuda(errorCode);

  std::cout << "STREAM triad of size " << std::setw(8) << N
            << " : min/avg/max: " << std::setw(11) << best << " "
            << std::setw(11) << avg / n_tests << " " << std::setw(11) << worst
            << " seconds or " << std::setw(8) << 1e-9 * 2 * N * N * N / best
            << " GFLOPS/s or " << std::setw(8)
            << 1e-9 * 3 * sizeof(float) * N / best << " GB/s" << std::endl;
}

int main(int argc, char **argv)
{
  if (argc % 2 == 0)
    {
      std::cout << "Error, expected odd number of common line arguments"
                << std::endl
                << "Expected line of the form" << std::endl
                << "-min 100 -max 1e8 -repeat -1" << std::endl;
      std::abort();
    }

  long N_min  = 8;
  long N_max  = -1;
  bool align  = false;
  long repeat = -1;
  // parse from the command line
  for (int l = 1; l < argc; l += 2)
    {
      std::string option = argv[l];
      if (option == "-min")
        N_min = static_cast<long>(std::stod(argv[l + 1]));
      else if (option == "-max")
        N_max = static_cast<long>(std::stod(argv[l + 1]));
      else if (option == "-repeat")
        repeat = std::atoll(argv[l + 1]);
      else if (option == "-align")
        align = std::atoi(argv[l + 1]);
      else
        std::cout << "Unknown option " << option << " - ignored!" << std::endl;
    }
  if (N_min < 1)
    {
      std::cout << "Expected positive size for min argument, got " << N_min
                << std::endl;
      return 0;
    }

  if (N_max < N_min)
    N_max = N_min;

  for (long n = N_min; n <= N_max; n = (1 + n * 1.1))
    {
      // round up to nearest multiple of 8
      n = (n + 7) / 8 * 8;
      benchmark_triad(align, n, repeat);
    }

  return 0;
}
