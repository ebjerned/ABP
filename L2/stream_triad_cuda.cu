

#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <vector>

#define AssertCuda(error_code) 									\
if(error_code != hipSuccess) 									\
{ 																\
	std::cout << "The cuda call in " << __FILE__ << " on line " \
	<< __LINE__ << " resulted in the error '" 					\
	<< hipGetErrorString(error_code) << "'" << std::endl; 		\
	std::abort();												\
}														 		\


const int block_size = 32;
const int chunk_size = 1;

__global__ void reduce0(int* g_idata, int* result){
	extern __shared__ int sdata[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	//printf("%d %d %d\n", tid, blockIdx.x, blockDim.x);
	sdata[tid] = g_idata[i];

	__syncthreads();

	for(unsigned int s=1; s<blockDim.x; s*=2){
		if(tid%(2*s)==0){
			sdata[tid] += sdata[tid+s];	
		}
		__syncthreads();
	}
	if(tid==0)
		atomicAdd(result, sdata[0]);
//		g_odata[blockIdx.x] = sdata[0];
	
}



__global__ void set_vector(const int N, const int val, int *x)
{
  const int idx_base = threadIdx.x + blockIdx.x * (blockDim.x * chunk_size);
  for (unsigned int i = 0; i < chunk_size; ++i)
    {
      const int idx = idx_base + i * block_size;
      if (idx < N)
        x[idx] = 1;
    }
}


// Run the actual benchmark
void benchmark_triad(const bool        align,
                     const std::size_t N,
                     const long long   repeat)
{
  int *v1;
  int* result;
  int res = 0;
  hipError_t error_code;
  // allocate memory on the device
  error_code = hipMalloc(&v1, N * sizeof(int));
  AssertCuda(error_code);
  //error_code = cudaMalloc(&v2, N * sizeof(int));
  //AssertCuda(error_code);
  error_code = hipMalloc(&result, sizeof(int));
  AssertCuda(error_code);
  const unsigned int n_blocks = (N + block_size - 1) / block_size;

  set_vector<<<n_blocks, block_size>>>(N, 17, v1);
  error_code = hipGetLastError();
  AssertCuda(error_code);
 /* 
  set_vector<<<n_blocks, block_size>>>(N, 0, v2);
  error_code = cudaGetLastError();
  AssertCuda(error_code);
  */

  std::vector<int> result_host(N);

  const unsigned            n_tests = 20;
  const unsigned long long int n_repeat =
    repeat > 0 ? repeat : std::max(1UL, 100000000U / N);
  double best = 1e10, worst = 0, avg = 0;
  for (unsigned int t = 0; t < n_tests; ++t)
    {
      // type of t1: std::chrono::steady_clock::time_point
      const auto t1 = std::chrono::steady_clock::now();

      for (unsigned int rep = 0; rep < n_repeat; ++rep){
	    error_code = hipMemset(result,0,sizeof(int));
		AssertCuda(error_code);
        reduce0<<<n_blocks, block_size, N>>>(v1, result); 
  		error_code = hipGetLastError();
  		AssertCuda(error_code);
		

	  }

      hipDeviceSynchronize();
      // measure the time by taking the difference between the time point
      // before starting and now
      const double time =
        std::chrono::duration_cast<std::chrono::duration<double>>(
          std::chrono::steady_clock::now() - t1)
          .count();

      best  = std::min(best, time / n_repeat);
      worst = std::max(worst, time / n_repeat);
      avg += time / n_repeat;
    }

  // Copy the result back to the host
  error_code = hipMemcpy(result_host.data(), v1, N*sizeof(int), hipMemcpyDeviceToHost);
  AssertCuda(error_code);
  error_code = hipMemcpy(&res, result, sizeof(int), hipMemcpyDeviceToHost);
  AssertCuda(error_code);
  std::cout << "Sum: " << res << std::endl;
  if (res != N)
    std::cout << "Error in computation, got "
              << (result_host[0] )<< " instead of 526"
              << std::endl;
  /*for(unsigned int i = 0; i < N; i++){
  	std::cout << result_host[i] << " ";
  }*/
  std::cout << "Finished printout" << std::endl;
  // Free the memory on the device
  hipFree(v1);
  //cudaFree(v2);
  //cudaFree(v3);
  hipFree(result);
  std::cout << "STREAM triad of size " << std::setw(8) << N
            << " : min/avg/max: " << std::setw(11) << best << " "
            << std::setw(11) << avg / n_tests << " " << std::setw(11) << worst
            << " seconds or " << std::setw(8) << 1e-6 * N / best
            << " MUPD/s or " << std::setw(8)
            << 1e-9 * 3 * sizeof(int) * N / best << " GB/s" << std::endl;
}

int main(int argc, char **argv)
{
  if (argc % 2 == 0)
    {
      std::cout << "Error, expected odd number of common line arguments"
                << std::endl
                << "Expected line of the form" << std::endl
                << "-min 100 -max 1e8 -repeat -1" << std::endl;
      std::abort();
    }

  long N_min  = 8;
  long N_max  = -1;
  bool align  = false;
  long repeat = -1;
  // parse from the command line
  for (int l = 1; l < argc; l += 2)
    {
      std::string option = argv[l];
      if (option == "-min")
        N_min = static_cast<long>(std::stod(argv[l + 1]));
      else if (option == "-max")
        N_max = static_cast<long>(std::stod(argv[l + 1]));
      else if (option == "-repeat")
        repeat = std::atoll(argv[l + 1]);
      else if (option == "-align")
        align = std::atoi(argv[l + 1]);
      else
        std::cout << "Unknown option " << option << " - ignored!" << std::endl;
    }
  if (N_min < 1)
    {
      std::cout << "Expected positive size for min argument, got " << N_min
                << std::endl;
      return 0;
    }

  if (N_max < N_min)
    N_max = N_min;

  for (long n = N_min; n <= N_max; n = (1 + n * 1.1))
    {
      // round up to nearest multiple of 8
      n = (n + 7) / 8 * 8;
      benchmark_triad(align, n, repeat);
    }

  return 0;
}

